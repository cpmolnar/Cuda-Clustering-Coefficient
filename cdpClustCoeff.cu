#include "hip/hip_runtime.h"
/**
 * Written by Carl Molnar
 * On November 3, 2016
 * CSCI 415 Assignment 2
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "readfile.h"
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>

using namespace std;

__device__ float
threadCalc(int *inputArr, int threadStartLoc, int lengthArr, int threadStartNode)
{
	int connections[4009];					//The array containing all connections to current node
   	float localSum = 0;					//The local sum from this node
  	int numConnections = 0;					//The counter of the number of connections on this node
  	int threadEndLoc = threadStartLoc + lengthArr;		//The end location of the thread in the input array

   	for(int i = threadStartLoc; i < threadEndLoc; i++){	//From the start of this node to the end,
       		if(inputArr[i] == 1){				//if this node has a connection in the input array,
        		connections[numConnections] = i;	//put it in the connections array
        		numConnections++;			//and increment the number of connections counter
       		}
       	}

   	int threadId = threadStartNode * lengthArr;		//The position of the thread in the input array
        int edges = 0;						//Counter for the number of edges
        int edgeTest1, edgeTest2, edgeCompare1, edgeCompare2;	//The variables for connections to compare

       	for(int i = 0; i < numConnections - 1; i++){						//From the start of this node's connections to the end, for the first connection,
           	for(int j = i + 1; j < numConnections; j++){					//From the start of this node's connections to the end, for the second connection,
                	edgeTest1 = connections[i] - threadId;					//Adjust each node for the position in the input array,
                	edgeTest2 = connections[j] - threadId;

                	edgeCompare1 = (edgeTest1 * lengthArr) + edgeTest2;			//And test both sides of the array for an edge
                	edgeCompare2 = (edgeTest2 * lengthArr) + edgeTest1;

              		if (inputArr[edgeCompare1] == 1 && inputArr[edgeCompare2] == 1)		//If there is an edge,
                	edges++;								//Increment the counter
        	}
        }

        if(numConnections < 2)								//If there are only two connections,
        	localSum = 0;								//These edges don't count
        else										//Otherwise,
	localSum = (float)(2 * edges) / (numConnections * (numConnections - 1));	//Do the calculation

        return localSum;								//And return the local sum
}

__global__ void kernelCalcCoeff( int *inputArr, int lengthArr,int threadPart, float *outputArr){

  	int threadStartLoc = threadIdx.x * lengthArr * threadPart; 	//The start location of the thread in the input array
	int threadStartNode = threadPart * threadIdx.x;			//The start location of the thread in the output array 
  	int threadEndNode = threadStartNode + threadPart;		//The end location of the thread in the output array
        __shared__ float localArr[4009];				//The output array with each node's number of edges
        float localSum;
	int i = 0;

  	if(i < lengthArr){										//If the first node is less than the length of the array, then
   		for(i = threadStartNode; i < threadEndNode; i++){    					//until we reach the last node the thread uses,
    			localSum = threadCalc(inputArr, threadStartLoc, lengthArr, i);			//get a local sum from the thread's current node
	 		outputArr[i] = localSum;							//and store it in the output array,
         		threadStartLoc += lengthArr;							//and then increment to the next node
  		}
	}
	outputArr = localArr;		//Assign the output array to the local array
}

int main(int argc, char **argv)
{
	readfile();
	int length = getNumNodes();

	size_t sizeSum = length * sizeof(float);
	float *h_S = (float *) malloc(sizeSum);

	int numElements = length * length;
	size_t sizeArray = numElements * sizeof(int);

	int *h_A = (int*) malloc(sizeArray);
	double Sum = 0.0;

	clock_t t1,t2;
   	h_A = getMatrix();

	hipError_t err = hipSuccess;

  	int  *d_A = NULL;
  	float *d_S = NULL;

   	int nBlocks = atoi(argv[1]);
   	int nThreads = atoi(argv[2]);
   	int partition = length/nThreads;

	//Allocate device memory for the input array
  	err = hipMalloc((int **)&d_A, sizeArray);
  	if (err != hipSuccess)
  	{
      		fprintf(stderr, "Failed to allocate device mat A (error code %s)!\n", hipGetErrorString(err));
      		exit(EXIT_FAILURE);
  	}

	//Allocate device memory for the output array
  	err = hipMalloc((float **)&d_S, sizeSum);
  	if (err != hipSuccess)
  	{
      		fprintf(stderr, "Failed to allocate device mat A (error code %s)!\n", hipGetErrorString(err));
      		exit(EXIT_FAILURE);
  	}

	//Copy the host result array to the device
 	err = hipMemcpy(d_S, h_S, sizeSum, hipMemcpyHostToDevice);
  	if (err != hipSuccess)
  	{
    		fprintf(stderr, "Failed to copy d_S from host to device (error code %s)!\n", hipGetErrorString(err));
      		exit(EXIT_FAILURE);
  	}

	//Copy the host input array to the device
  	err = hipMemcpy(d_A, h_A, sizeArray, hipMemcpyHostToDevice);
  	if (err != hipSuccess)
  	{
    		fprintf(stderr, "Failed to copy mat A 1 from host to device (error code %s)!\n", hipGetErrorString(err));
      		exit(EXIT_FAILURE);
  	}

	cout<<"Device memory allocated. Starting threads..."<<endl;

	//Start the timer
	t1=clock();
	//Launch the Cuda kernel
	kernelCalcCoeff<<<nBlocks,nThreads>>>(d_A, length, partition, d_S);
 
	//End the timer
	t2=clock();

	cout<<"Threads complete. Finding sum..."<<endl;

	//Copy the device result array to the host
	hipMemcpy(h_S, d_S, sizeSum, hipMemcpyDeviceToHost);

	if(err != hipSuccess)
	{
  		fprintf(stderr,"Failed to copy mat 3 A from device to host(error code %s)!\n",hipGetErrorString(err));
  		exit(EXIT_FAILURE);
	}

	//Free the device memory
	err = hipFree(d_A);
	err = hipFree(d_S);
	
	//Sum up the partial sums of the threads from the output array
	for(int t = 0; t < length; t++){
    		Sum += h_S[t];
  	}

  	printf("=====================================================\n");
  	double finalSum = (Sum/length);
  	cout<<"Clustering coefficient: "<<finalSum<<endl;

	//Free the host memory
  	free(h_A);
  	free(h_S);

	//Reset the device
    	err = hipDeviceReset();
      	if (err != hipSuccess)
      	{
          	fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
          	exit(EXIT_FAILURE);
      	}

	//Find the time and print it
       	float timeElapsed = ((float)t2-(float)t1);  
       	float timeInSeconds = timeElapsed / CLOCKS_PER_SEC;
       	cout<<"Time: "<<timeInSeconds<<" seconds"<<endl;

  	return 0;
}
